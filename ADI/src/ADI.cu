#include "hip/hip_runtime.h"
/*
 * ADI.cu
 *
 *  Created on: 22 Oct 2013
 *      Author: geekity
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <algorithm>

#include "ADI.h"
#include "constants/constants.h"
#include "constants/alloc.h"

using namespace std;

#define TESTING
#define TOLL 1e-4
#define CHUNK_MAX 256
//#define AN (-Q_E*DT/H*DT/H/2)

/* ADI class public methods */

/* Constructors */
ADI::ADI(int N_tmp, int S_tmp) {
	S = S_tmp;
	N = N_tmp;
	pcr = new PCR(N, S);
	old_err = sqrt(-1);

	h_phi_new = (TYPE_VAR*) safe_malloc(N*S*sizeof(TYPE_VAR));
	h_arr = (TYPE_VAR*) safe_malloc(N*S*sizeof(TYPE_VAR));
	check_return(hipMalloc((TYPE_VAR**)&d_phi_new, N*S*sizeof(TYPE_VAR)));
	check_return(hipMalloc((TYPE_VAR**)&d_phi_bar, N*S*sizeof(TYPE_VAR)));
	check_return(hipMalloc((TYPE_VAR**)&d_u, N*S*sizeof(TYPE_VAR)));
	check_return(hipMalloc((TYPE_VAR**)&phi_trans, N*S*sizeof(TYPE_VAR)));
	check_return(hipMalloc((TYPE_VAR**)&rho_trans, N*S*sizeof(TYPE_VAR)));
}

/* Destructor */
ADI::~ADI() {
	delete pcr;

	safe_free(h_phi_new);
	safe_free(h_arr);

	check_return(hipFree(d_phi_new));
	check_return(hipFree(d_phi_bar));
	check_return(hipFree(d_u));
	check_return(hipFree(phi_trans));
	check_return(hipFree(rho_trans));
}

/* Actual DADI solver implementation */
__host__ void ADI::adi_solver(TYPE_VAR* d_phi, TYPE_VAR* d_rho) {
	/**/
	TYPE_VAR dt = 1.0;
	TYPE_VAR dh1 = 1.0;
	TYPE_VAR dh2 = 1.0;
	bool accept = false;	/* bool to determine whether iteration was accepted */

	/* finds the transpose of rho for building up of PCR solver RHS */
	transpose<<<BLOCKS, THREADS>>>(d_rho, rho_trans, N, S);
	hipDeviceSynchronize();
	check_return(hipGetLastError());

	/* loops until convergence is achieved */
	do {
		if (accept) {
			check_return(hipMemcpy(d_phi, h_phi_new, N*S*sizeof(TYPE_VAR), hipMemcpyHostToDevice));
		}

		check_return(hipMemcpy(d_phi_new, d_phi, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToDevice));
		check_return(hipMemcpy(d_phi_bar, d_phi, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToDevice));

		/* 2 double sweeps of 1*dt */
		double_sweep(d_phi_new, d_rho, dt, dh1, dh2);
		double_sweep(d_phi_new, d_rho, dt, dh1, dh2);

		check_return(hipMemcpy(h_phi_new, d_phi_new, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));

		/* 1 double sweep of 2*dt */
		double_sweep(d_phi_bar, d_rho, 2*dt, dh1, dh2);

	} while (check_err(d_phi, d_rho, &dt, &accept, dh1, dh2));

	check_return(hipMemcpy(d_phi, h_phi_new, N*S*sizeof(TYPE_VAR), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

/*	ADI_rescale<<<S, N>>>(d_phi, EPSILON0, N, S);
	hipDeviceSynchronize();
	check_return(hipGetLastError());*/
}

/* ADI class private methods */

/* Check PCR matrix seeding from kerne functions */
void ADI::check_arrays() {
	TYPE_VAR* A1 = (TYPE_VAR*) safe_malloc(N*S*sizeof(TYPE_VAR));
	TYPE_VAR* A2 = (TYPE_VAR*) safe_malloc(N*S*sizeof(TYPE_VAR));
	TYPE_VAR* A3 = (TYPE_VAR*) safe_malloc(N*S*sizeof(TYPE_VAR));
	TYPE_VAR* B = (TYPE_VAR*) safe_malloc(N*S*sizeof(TYPE_VAR));

	check_return(hipMemcpy(A1, pcr->A1_arr(), N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));
	check_return(hipMemcpy(A2, pcr->A2_arr(), N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));
	check_return(hipMemcpy(A3, pcr->A3_arr(), N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));
	check_return(hipMemcpy(B, pcr->B_arr(), N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));

	cout << "A1: " << endl;
	for (int i = 0; i < S; i++) {
		for (int j = 0; j < N; j++) {
			cout << A1[i*N+j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	cout << "A2: " << endl;
	for (int i = 0; i < S; i++) {
		for (int j = 0; j < N; j++) {
			cout << A2[i*N+j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	cout << "A3: " << endl;
	for (int i = 0; i < S; i++) {
		for (int j = 0; j < N; j++) {
			cout << A3[i*N+j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	cout << "B: " << endl;
	for (int i = 0; i < S; i++) {
		for (int j = 0; j < N; j++) {
			cout << B[i*N+j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	safe_free(A1);
	safe_free(A2);
	safe_free(A3);
	safe_free(B);
}

/* Checks solution convergence and adjusts dt */
bool ADI::check_err(TYPE_VAR* d_phi, TYPE_VAR* rho, TYPE_VAR* dt, bool* accept,
	TYPE_VAR dh1, TYPE_VAR dh2) {
	check_return(hipMemcpy(d_u, d_phi, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToDevice));
	calc_dif_iter<<<S, N>>>(d_phi_new, d_u, d_phi_bar, N, S);

	hipDeviceSynchronize();
	check_return(hipGetLastError());

	TYPE_VAR tp_top = my_reduction(d_phi_bar);
	TYPE_VAR tp_bottom = my_reduction(d_phi_new);

//	check_return(hipMemcpy(d_u, rho, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToDevice));
//	check_return(hipMemcpy(d_phi_new, h_phi_new, N*S*sizeof(TYPE_VAR), hipMemcpyHostToDevice));

//	ADI_converge<<<S, N>>>(d_phi_new, d_u, N, S, dh1, dh2);
	TYPE_VAR tp_u = my_reduction(d_u);

	tp_top = sqrt(tp_top);
	tp_bottom = sqrt(tp_bottom);
	tp_u = sqrt(tp_u);

	assert(tp_top==tp_top);
	assert(tp_bottom==tp_bottom);
	assert(tp_u==tp_u);

	if (tp_u == old_err) {
		cout << "Error! You have exceeded the accuracy of the solver and caused an overflow!" << endl;
		exit(1);
	} else {
		old_err = tp_u;
	}

	cout << "tp_u: " << tp_u << endl;

	if (tp_u < TOLL) return false;

	TYPE_VAR tp = tp_top/tp_bottom;
	if (tp <= 0.05) {
		*dt *= 4;
		*accept = true;
	} else if (tp <= 0.1) {
		*dt *= 2;
		*accept = true;
	} else if (tp <= 0.3) {
		*dt *= sqrt(5.0);
		*accept = true;
	} else if (tp <= 0.4) {
		*dt *= 0.5;
		*accept = true;
	} else if (tp <= 0.6) {
		*dt *= 0.25;
		*accept = true;
	} else {
		*dt *= 0.0625;
		*accept = false;
	}
	return true;
}

/* Single double sweep of ADI solver. Input rho & phi are expected to be row sorted */
/* (i.e. x = column, y = row, index_A = y*N + x)                                    */
void ADI::double_sweep(TYPE_VAR* phi_new, TYPE_VAR* rho, TYPE_VAR dt,
	TYPE_VAR dh1, TYPE_VAR dh2) {

	/* system solved along vertical */
	pcr->ADI_flip(S, N);	// sets up PCR # of equations and # of systems

	// Sets up matrix A and vector B for PCR solver
	calcAB<<<N, S>>>(pcr->A1_arr(), pcr->A2_arr(), pcr->A3_arr(), pcr->B_arr(),
			phi_new, rho_trans, dt, dh2, dh1, S, N);
	hipDeviceSynchronize();
	check_return(hipGetLastError());

	// solve the mesh system
	pcr->PCR_solve(phi_new);

	// transpose phi for setting up vector B for next sweep
	transposes(phi_new);

	/* system solved along horizontal */
	pcr->ADI_flip(N, S);	// sets up PCR # of equations and # of systems

	// Sets up matrix A and vector B for PCR solver
	calcAB<<<S, N>>>(pcr->A1_arr(), pcr->A2_arr(), pcr->A3_arr(), pcr->B_arr(),
			phi_new, rho, dt, dh1, dh2, N, S);
	hipDeviceSynchronize();
	check_return(hipGetLastError());

	// solve the mesh system
	pcr->PCR_solve(phi_new);

	// transpose phi to return to original orientation
	transposes(phi_new);
}

/* transposes phi "in place" (not really in place, just a wrapper method) */
void ADI::transposes(TYPE_VAR* phi_new) {
	transpose<<<BLOCKS, THREADS>>>(phi_new, phi_trans, N, S);
	hipDeviceSynchronize();
	check_return(hipGetLastError());

	check_return(hipMemcpy(phi_new, phi_trans, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToDevice));
}

/* Sum an array */
TYPE_VAR ADI::my_reduction(TYPE_VAR* d_arr)	{
	int B = (N*S+CHUNK_MAX-1)/CHUNK_MAX;
	shared_reduction<<<B, CHUNK_MAX>>>(d_arr, N*S);

	hipDeviceSynchronize();
	check_return(hipGetLastError());

	check_return(hipMemcpy(h_arr, d_arr, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));

	for (int i = CHUNK_MAX; i < S*N; i += CHUNK_MAX) {
		h_arr[0] += h_arr[i];
	}
	return h_arr[0];
}

/* assert array does not consist of NaN */
void ADI::assert_notnan(TYPE_VAR* d_arr) {
	check_return(hipMemcpy(h_arr, d_arr, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));
	for (int i = 0; i < N*S; i++) {
		assert(h_arr[i] == h_arr[i]);
	}
}


/* Device functions */

/* Set up the A matrix and B vector for 2 dimensional Poisson equation */
__global__ void calcAB(TYPE_VAR* A1, TYPE_VAR* A2, TYPE_VAR* A3, TYPE_VAR* B, TYPE_VAR* phi,
	TYPE_VAR* rho, TYPE_VAR dt, TYPE_VAR dh1, TYPE_VAR dh2, int N, int S) {

	int tid1 = blockIdx.x*N + threadIdx.x;

	if (tid1 < N*S) {
		A1[tid1] = (threadIdx.x == 0) ? 0.0 : -dt/(dh1*dh1);
		A2[tid1] = 1 + 2*dt/(dh1*dh1) + 2*dt/(dh2*dh2);
		A3[tid1] = (threadIdx.x == N-1) ? 0.0 : -dt/(dh1*dh1);
		int tid2 = threadIdx.x*S + blockIdx.x;
		B[tid1] = rho[tid1]*dt + phi[tid2];
		B[tid1] += (blockIdx.x == 0) ? 0.0 : phi[tid2-1]*dt/(dh2*dh2);
		B[tid1] += (blockIdx.x == S-1) ? 0.0 : phi[tid2+1]*dt/(dh2*dh2);
	}
}

/* Calculate new B vector only (case if dt is constant) */
__global__ void recalcB(TYPE_VAR* B, TYPE_VAR* phi, TYPE_VAR* rho, TYPE_VAR dt, TYPE_VAR dh1,
	TYPE_VAR dh2, int N, int S) {
	int tid1 = blockIdx.x*N + threadIdx.x;
	if (tid1 < N*S) {
		int tid2 = threadIdx.x*S + blockIdx.x;
		B[tid1] = rho[tid1]*dt + phi[tid2];
		B[tid1] += (blockIdx.x == 0) ? 0.0 : phi[tid2-1]*dt/(dh2*dh2);
		B[tid1] += (blockIdx.x == S-1) ? 0.0 : phi[tid2+1]*dt/(dh2*dh2);
	}
}

/* Check difference between iterations */
__global__ void calc_dif_iter(TYPE_VAR* phi_new, TYPE_VAR* phi_old, TYPE_VAR* phi_bar,
	int N, int S) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if (tid < N*S) {
		phi_bar[tid] -= phi_new[tid];
		phi_new[tid] -= phi_old[tid];
		phi_old[tid] = (phi_old[tid] == 0.0) ? phi_new[tid] : phi_new[tid]/phi_old[tid];

		phi_bar[tid] *= phi_bar[tid];
		phi_new[tid] *= phi_new[tid];
		phi_old[tid] *= phi_old[tid];
	}
}

/* Matrix transpose (see G. Reutsch, P. Micikevicius, Optimizing matrix transpose in CUDA) */
__global__ void transpose(TYPE_VAR *iden, TYPE_VAR *oden, int N, int S) {

	// Dimension of tile adjusted to avoid bank conflicts
	__shared__ TYPE_VAR tile[TILE_WIDTH][TILE_WIDTH+1];

	int blockIdx_x, blockIdx_y;

	// Use coordinates in diagonal context to avoid partition camping
	if (N == S) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x+blockIdx.y) % gridDim.x;
	} else {
		int bid = blockIdx.x + gridDim.x*blockIdx.y;
		blockIdx_y = bid % gridDim.y;
		blockIdx_x = ((bid/gridDim.y)+blockIdx_y) % gridDim.x;
	}

	int tidx = threadIdx.x + blockIdx_x*TILE_WIDTH;
	int tidy = threadIdx.y + blockIdx_y*TILE_WIDTH;
	int index_in = tidx + tidy*N;

	// copy data to tile
	for (int i = 0; i < (TILE_WIDTH+SHARE_Y-1); i += SHARE_Y) {
		for (int j = 0; j < (TILE_WIDTH+SHARE_X-1); j += SHARE_X) {
			if (((tidx+j < N)
					&& (tidy+i < S))
					&& ((i+threadIdx.y < TILE_WIDTH)
					&& (j+threadIdx.x < TILE_WIDTH))) {
				tile[threadIdx.y+i][threadIdx.x+j] = iden[index_in + j + i*N];
			}
		}
	}

	__syncthreads();

	tidx = threadIdx.x + blockIdx_y*TILE_WIDTH;
	tidy = threadIdx.y + blockIdx_x*TILE_WIDTH;
	int index_out = tidx + tidy*S;

	// insert transpose into output array
	for (int i = 0; i < (TILE_WIDTH+SHARE_Y-1); i += SHARE_Y) {
		for (int j = 0; j < (TILE_WIDTH+SHARE_X-1); j += SHARE_X) {
			if (((tidx+j < S)
					&& (tidy+i < N))
					&& ((i+threadIdx.y < TILE_WIDTH)
					&& (j+threadIdx.x < TILE_WIDTH))) {
				oden[index_out + j + i*S] = tile[threadIdx.x+j][threadIdx.y+i];
			}
		}
	}
}

/* partial reduction in shared memory */
__global__ void shared_reduction(TYPE_VAR* arr, int size) {
	__shared__ TYPE_VAR a[CHUNK_MAX];
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if (tid < size) {
		a[threadIdx.x] = arr[tid];

		__syncthreads();

		int step = CHUNK_MAX/2;

		while (step > 0) {
			if ((threadIdx.x < step) && (tid+step < size))
				a[threadIdx.x] += a[threadIdx.x+step];
			__syncthreads();
			step /= 2;
		}

		if (threadIdx.x == 0) arr[tid] = a[threadIdx.x];
		else arr[tid] = 0.0;
	}
}

/* check for convergence */
__global__ void ADI_converge(TYPE_VAR* phi, TYPE_VAR* rho, int N, int S, TYPE_VAR dh1, TYPE_VAR dh2) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < N*S) {
		TYPE_VAR dif1 = (threadIdx.x == 0) ? -2*phi[tid] : -2*phi[tid] + phi[tid-1];
		dif1 += (threadIdx.x == N-1) ? 0 : phi[tid+1];
		dif1 /= (dh1*dh1);
		TYPE_VAR dif2 = (blockIdx.x == 0) ? -2*phi[tid] : -2*phi[tid] + phi[tid-N];
		dif2 += (blockIdx.x == S-1) ? 0 : phi[tid+N];
		dif2 /= (dh2*dh2);

		rho[tid] = rho[tid] + dif1 + dif2;
		rho[tid] *= rho[tid];
	}
}

/* Rescale */
__global__ void ADI_rescale(TYPE_VAR* phi, TYPE_VAR scale, int N, int S) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < N*S) {
		phi[tid] /= scale;
	}
}

/* Test function to see how well solution satisfies equation */
void ADI_test(TYPE_VAR* phi, TYPE_VAR* rho, int N, int S, TYPE_VAR dh1, TYPE_VAR dh2) {

	for (int i = 0; i < S; i++) {
		for (int j = 0; j < N; j++) {
			TYPE_VAR dif1 = (j == 0) ? -2*phi[i*N+j] : -2*phi[i*N+j] + phi[i*N+j-1];
			dif1 += (j == N-1) ? 0 : phi[i*N+j+1];
			dif1 /= (dh1*dh1);
			TYPE_VAR dif2 = (i == 0) ? -2*phi[i*N+j] : -2*phi[i*N+j] + phi[i*N+j-N];
			dif2 += (i == S-1) ? 0 : phi[i*N+j+N];
			dif2 /= (dh2*dh2);

			rho[i*N+j] = rho[i*N+j] + dif1 + dif2;
		}
	}
}

/* test main */

#ifdef TESTING

int main() {
	cout << "Hello World!" << endl;

	int N = 63;
	int S = 63;
	ADI* adi = new ADI(N, S);

	TYPE_VAR phi[N*S];
	TYPE_VAR rho[N*S];
	for (int i = 0; i < N*S; i++) {
		phi[i] = 0.0;
		rho[i] = 1.6e-4*(H*H)/EPSILON0;
//		rho[i] = 0;
	}

	TYPE_VAR* d_phi;
	TYPE_VAR* d_rho;

	check_return(hipMalloc((TYPE_VAR**)&d_phi, N*S*sizeof(TYPE_VAR)));
	check_return(hipMalloc((TYPE_VAR**)&d_rho, N*S*sizeof(TYPE_VAR)));

	check_return(hipMemcpy(d_phi, phi, N*S*sizeof(TYPE_VAR), hipMemcpyHostToDevice));
	check_return(hipMemcpy(d_rho, rho, N*S*sizeof(TYPE_VAR), hipMemcpyHostToDevice));

	adi->adi_solver(d_phi, d_rho);

	check_return(hipMemcpy(phi, d_phi, N*S*sizeof(TYPE_VAR), hipMemcpyDeviceToHost));

	for (int i = 0; i < S; i++) {
		for (int j = 0; j < N; j++) {
//			phi[i*N+j] *= Q_E*DENSITY/512*(H*H)/EPSILON0;
			cout << phi[i*N+j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	ADI_test(phi, rho, N, S, 1.0, 1.0);
	TYPE_VAR e_tot = 0.0;
	for (int i = 0; i < S; i++) {
		for (int j = 0; j < N; j++) {
			e_tot += rho[i*N+j]*rho[i*N+j];
		}
	}
	e_tot = sqrt(e_tot);
	cout << "err: " << e_tot << endl;

	hipFree(d_phi);
	hipFree(d_rho);

	delete adi;

	return 0;
}

#endif
